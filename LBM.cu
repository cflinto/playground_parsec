#include "hip/hip_runtime.h"
extern "C" {
#include "LBM.cuh"
}

void d2q9_initial_value_d_caller(Grid grid, double *subgrid, int subgridX, int subgridY, int d)
{
    d2q9_initial_value_d<<<256, 256>>>(grid, subgrid, subgridX, subgridY, d);
}

void d2q9_save_reduce_caller(Grid grid, double *base_subgrid, double *reduced_subgrid, int subgridX, int subgridY, int d)
{
    d2q9_save_reduce<<<256, 256>>>(grid, base_subgrid, reduced_subgrid, subgridX, subgridY, d);
}

void d2q9_LBM_step_caller(Grid grid,
                double **subgrid_FROM_D,
                double **subgrid_TO_D,
                int horizontal_uncomputed_number, int vertical_uncomputed_number,
                bool has_from_interface_horizontal,
                bool has_from_interface_vertical,
                bool has_to_interface_horizontal,
                bool has_to_interface_vertical,
                double *interface_left, double *interface_right,
                double *interface_down, double *interface_up,
                int subgridX, int subgridY)
{
    // wrap the arrays
    SubgridArray subgrid_FROM_D_wrapped;
    SubgridArray subgrid_TO_D_wrapped;
    for(int d=0;d<grid.directionsNumber;d++)
    {
        subgrid_FROM_D_wrapped.subgrid[d] = subgrid_FROM_D[d];
        subgrid_TO_D_wrapped.subgrid[d] = subgrid_TO_D[d];
    }

    printf("d2q9_LBM_step_caller: has_from_interface_horizontal=%d, has_from_interface_vertical=%d, has_to_interface_horizontal=%d, has_to_interface_vertical=%d\n",
        has_from_interface_horizontal, has_from_interface_vertical, has_to_interface_horizontal, has_to_interface_vertical);

    d2q9_LBM_step<<<256, 256>>>(grid,
                subgrid_FROM_D_wrapped,
                subgrid_TO_D_wrapped,
                horizontal_uncomputed_number, vertical_uncomputed_number,
                has_from_interface_horizontal,
                has_from_interface_vertical,
                has_to_interface_horizontal,
                has_to_interface_vertical,
                interface_left, interface_right,
                interface_down, interface_up,
                subgridX, subgridY);
}



__device__
int get_dir(int i, int j)
{
    const int dirs[9][2] = {
        {-1, -1},
        {0, -1},
        {1, -1},
        {-1, 0},
        {0, 0},
        {1, 0},
        {-1, 1},
        {0, 1},
        {1, 1}
    };

    return dirs[i][j];
}

__device__
void fluid_to_kin(const double *w, double *f)
{
    static const double c2 = 1. / 3.;
    double dotvel = 0, vel2 = 0, l2 = 0, l4 = 0, c4 = 0;

    l2 = MAXIMUM_VELOCITY * MAXIMUM_VELOCITY;
    double l2_ov_c2 = l2 / c2;

    l4 = l2 * l2;
    c4 = c2 * c2;

    vel2 = (w[1] * w[1] + w[2] * w[2]) / (w[0] * w[0]);
    dotvel = sqrt(l2) * (get_dir(4,0) * w[1] + get_dir(4,1) * w[2]) / w[0];

    f[4] = (4. / 9.) * w[0] *
    (1.0 + (l2_ov_c2)*dotvel + l4 / (2. * c4) * dotvel * dotvel -
    l2 / (2. * c2) * vel2);

    // perpendicular directions
    for (size_t i = 1; i < 9; i+=2) {
        dotvel = sqrt(l2) * (get_dir(i,0) * w[1] + get_dir(i,1) * w[2]) / w[0];
        f[i] = (1. / 9.) * w[0] *
        (1.0 + (l2_ov_c2)*dotvel + l4 / (2. * c4) * dotvel * dotvel -
        l2 / (2. * c2) * vel2);
    }
    // diagonal directions
    for (size_t it = 0; it < 4; it++) {
        size_t i = it * 2 + 2*(it>1);
        dotvel = sqrt(l2) * (get_dir(i,0) * w[1] + get_dir(i,1) * w[2]) / w[0];
        f[i] = (1. / 36.) * w[0] *
        (1.0 + (l2_ov_c2)*dotvel + l4 / (2. * c4) * dotvel * dotvel -
        l2 / (2. * c2) * vel2);
    }
}

__device__
void kin_to_fluid(const double *f, double *w)
{
    w[0] = 0;
    w[1] = 0;
    w[2] = 0;

    for (int i = 0; i < 9; i++) {
        w[0] = w[0] + f[i];
        w[1] = w[1] + MAXIMUM_VELOCITY * get_dir(i,0) * f[i];
        w[2] = w[2] + MAXIMUM_VELOCITY * get_dir(i,1) * f[i];
    }
}


__device__
void d2q9_t0(double w[3], double x, double y)
{
    double rho, u, v;
    rho = 1;
    u = 0.03;
    v = 0.00001; // to get instability

    if ((x - CYLINDER_CENTER_X) * (x - CYLINDER_CENTER_X) + (y - CYLINDER_CENTER_Y) * (y - CYLINDER_CENTER_Y) < CYLINDER_RADIUS * CYLINDER_RADIUS) {
        u = 0;
        v = 0;
    }

    w[0] = rho;
    w[1] = rho * u;
    w[2] = rho * v;

    // w[0] = rho;
    // w[1] = rho * u;
    // w[2] = rho * v;
}


// Initialize all the values including the ghost cells
__global__
void d2q9_initial_value_d(Grid grid, double *subgrid, int subgridX, int subgridY, int d)
{
    (void)d; // same for all directions

    int stride = blockDim.x * gridDim.x;

	int cellNum = grid.subgridTrueSize[0] * grid.subgridTrueSize[1];

	for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < cellNum; id += stride)
	{
        int true_x = id % grid.subgridTrueSize[0];
        int true_y = id / grid.subgridTrueSize[0];

        int logical_x = true_x - grid.overlapSize[0];
        int logical_y = true_y - grid.overlapSize[1];

        int xgInt = logical_x + subgridX * grid.subgridOwnedSize[0];
        int ygInt = logical_y + subgridY * grid.subgridOwnedSize[1];

        double xg = grid.physicalMinCoords[0] + (((double)xgInt + 0.5) * grid.physicalSize[0] / (double)grid.size[0]);
        double yg = grid.physicalMinCoords[1] + (((double)ygInt + 0.5) * grid.physicalSize[1] / (double)grid.size[1]);

        double w[3];
        d2q9_t0(w, xg, yg);
        double f[3][3];
        fluid_to_kin(w, &f[0][0]);

        for(int c=0;c<grid.conservativesNumber;++c)
        {
            int subgrid_id = c * cellNum + id;
            subgrid[subgrid_id] = f[d][c];
        }
    }
}

// kernel used for the reduction for the save
// agregates the values in the stencil by doing an average
__global__
void d2q9_save_reduce(Grid grid, double *base_subgrid, double *reduced_subgrid, int subgridX, int subgridY, int d)
{
    int stride = blockDim.x * gridDim.x;

    int stencil_num_x_in_subgrid = grid.subgridOwnedSize[0] / grid.saveStencilSize[0];
    int stencil_num_y_in_subgrid = grid.subgridOwnedSize[1] / grid.saveStencilSize[1];

    assert(stencil_num_x_in_subgrid * grid.saveStencilSize[0] == grid.subgridOwnedSize[0]);
    assert(stencil_num_y_in_subgrid * grid.saveStencilSize[1] == grid.subgridOwnedSize[1]);

    int cellNum = stencil_num_x_in_subgrid * stencil_num_y_in_subgrid * grid.conservativesNumber;

    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < cellNum; id += stride)
    {
        int local_stencil_x = id % stencil_num_x_in_subgrid;
        int local_stencil_y = (id / stencil_num_x_in_subgrid) % stencil_num_y_in_subgrid;
        int c = id / (stencil_num_x_in_subgrid * stencil_num_y_in_subgrid);

        int global_stencil_x = local_stencil_x + subgridX * stencil_num_x_in_subgrid;
        int global_stencil_y = local_stencil_y + subgridY * stencil_num_y_in_subgrid;

        double average = 0;

        for (int i = 0; i < grid.saveStencilSize[0]; i++) {
            for (int j = 0; j < grid.saveStencilSize[1]; j++) {
                int true_x = local_stencil_x * grid.saveStencilSize[0] + i + grid.overlapSize[0];
                int true_y = local_stencil_y * grid.saveStencilSize[1] + j + grid.overlapSize[1];

                int base_subgrid_id = c * grid.subgridTrueSize[0] * grid.subgridTrueSize[1] + true_y * grid.subgridTrueSize[0] + true_x;

                average += base_subgrid[base_subgrid_id];
            }
        }

        average /= grid.saveStencilSize[0] * grid.saveStencilSize[1];

        int reduced_subgrid_id =
                        d * grid.sizeOfSavedData[0] * grid.sizeOfSavedData[1] * grid.conservativesNumber
                        + c * grid.sizeOfSavedData[0] * grid.sizeOfSavedData[1]
                        + global_stencil_y * grid.sizeOfSavedData[0]
                        + global_stencil_x;

        reduced_subgrid[reduced_subgrid_id] = average;
    }
}



// void d2q9_step(d2q9 *lbm) {
//   d2q9_shift(lbm);
//   d2q9_relax(lbm);
//   d2q9_boundary(lbm);
// }

// void d2q9_relax(d2q9 *lbm) {

// #pragma omp for schedule(static) nowait
//   for (size_t i = 0; i < lbm->nx; i++) {
//     for (size_t j = 0; j < lbm->ny; j++) {
//       double f[9];
//       double feq[9];
//       for (size_t k = 0; k < 9; k++) {
//         f[k] = lbm->fnext[k][i * lbm->ny + j];
//       }
//       double w[3];
//       kin_to_fluid(f, w, lbm);
//       for (size_t k = 0; k < 3; k++) {
//         lbm->w[k][i * lbm->ny + j] = w[k];
//         // printf("u=%f\n",w[1]/w[0]);
//       }
//       fluid_to_kin(w, feq, lbm);
//       for (size_t k = 0; k < 9; k++) {
//         lbm->f[k][i * lbm->ny + j] =
// 	  _RELAX * feq[k] + (1 - _RELAX) * lbm->fnext[k][i * lbm->ny + j];
//       }
//     }
//   }
// }

// void d2q9_boundary(d2q9 *lbm) {

// #pragma omp for schedule(static) nowait
//   for (size_t i = 0; i < lbm->nx; i++) {
//     for (size_t j = 0; j < lbm->ny; j++) {
//       double x = i * lbm->dx;
//       double y = j * lbm->dx;
//       if (mask(x, y)) {
//         double wb[3];
//         imposed_data(x, y, lbm->tnow, wb);
//         double fb[9];
//         fluid_to_kin(wb, fb, lbm);
//         for (size_t k = 0; k < 9; k++) {
//           lbm->f[k][i * lbm->ny + j] =
// 	    _RELAX * fb[k] + (1 - _RELAX) * lbm->f[k][i * lbm->ny + j];
//         }
//       }
//     }
//   }
// }

// void d2q9_solve(d2q9 *lbm, double tmax) {
//   lbm->tmax = tmax;
//   double local_tnow = lbm->tnow;
//   const double dt = lbm->dx / lbm->smax;
//   const size_t num_iter = (size_t)(ceil((tmax - local_tnow) / dt));
//   const size_t inter_print = num_iter / 10 == 0 ? 1 : num_iter / 10;
// #ifdef _OPENMP
//   double tstart_chunk;
//   tstart_chunk = omp_get_wtime();
// #endif
// #pragma omp parallel default(none) shared(lbm, tmax, tstart_chunk, dt, inter_print)	\
//   firstprivate(local_tnow)
//   {
//     size_t iter_count = 0;

//     while (local_tnow < tmax) {
//       d2q9_step(lbm);
// #pragma omp single nowait
//       lbm->tnow += dt;
//       if (!iter_count) {
// #ifdef _OPENMP
// #pragma omp master
//         {
//           double tend_chunk = omp_get_wtime();
//           printf("t=%f dt=%f tmax=%f (%zu iter in %.3fs)\n", lbm->tnow, dt,
//                  lbm->tmax, inter_print, tend_chunk - tstart_chunk);
//           tstart_chunk = tend_chunk;
//         }
// #else
//         printf("t=%f dt=%f tmax=%f\n", lbm->tnow, dt, lbm->tmax);
// #endif
//       }
// #pragma omp barrier
//       iter_count = iter_count == inter_print ? 0 : iter_count + 1;
//       local_tnow += dt;
//     }
//   }
// }

// void fluid_to_kin(const double *w, double *f, d2q9 *lbm) {
//   static const double c2 = 1. / 3.;
//   double dotvel = 0, vel2 = 0, l2 = 0, l4 = 0, c4 = 0;

//   l2 = lbm->smax * lbm->smax;
//   double l2_ov_c2 = l2 / c2;

//   l4 = l2 * l2;
//   c4 = c2 * c2;

//   vel2 = (w[1] * w[1] + w[2] * w[2]) / (w[0] * w[0]);
//   dotvel = sqrt(l2) * (lbm->vel[0][0] * w[1] + lbm->vel[0][1] * w[2]) / w[0];

//   f[0] = (4. / 9.) * w[0] *
//     (1.0 + (l2_ov_c2)*dotvel + l4 / (2. * c4) * dotvel * dotvel -
//      l2 / (2. * c2) * vel2);

//   for (size_t i = 1; i < 5; i++) {
//     dotvel = sqrt(l2) * (lbm->vel[i][0] * w[1] + lbm->vel[i][1] * w[2]) / w[0];
//     f[i] = (1. / 9.) * w[0] *
//       (1.0 + (l2_ov_c2)*dotvel + l4 / (2. * c4) * dotvel * dotvel -
//        l2 / (2. * c2) * vel2);
//   }
//   for (size_t i = 5; i < 9; i++) {
//     dotvel = sqrt(l2) * (lbm->vel[i][0] * w[1] + lbm->vel[i][1] * w[2]) / w[0];
//     f[i] = (1. / 36.) * w[0] *
//       (1.0 + (l2_ov_c2)*dotvel + l4 / (2. * c4) * dotvel * dotvel -
//        l2 / (2. * c2) * vel2);
//   }
// }

// void kin_to_fluid(const double *restrict f, double *restrict w, d2q9 *lbm) {

//   w[0] = 0;
//   w[1] = 0;
//   w[2] = 0;
//   double c = lbm->smax;

//   for (size_t i = 0; i < 9; i++) {
//     w[0] = w[0] + f[i];
//     w[1] = w[1] + c * lbm->vel[i][0] * f[i];
//     w[2] = w[2] + c * lbm->vel[i][1] * f[i];
//   }
// }



//   lbm->vel[0][0] = 0;
//   lbm->vel[0][1] = 0;

//   lbm->vel[1][0] = 1;
//   lbm->vel[1][1] = 0;

//   lbm->vel[2][0] = 0;
//   lbm->vel[2][1] = 1;

//   lbm->vel[3][0] = -1;
//   lbm->vel[3][1] = 0;

//   lbm->vel[4][0] = 0;
//   lbm->vel[4][1] = -1;

//   lbm->vel[5][0] = 1;
//   lbm->vel[5][1] = 1;

//   lbm->vel[6][0] = -1;
//   lbm->vel[6][1] = 1;

//   lbm->vel[7][0] = -1;
//   lbm->vel[7][1] = -1;

//   lbm->vel[8][0] = 1;
//   lbm->vel[8][1] = -1;



// similar to D2Q9_step
__global__
void d2q9_LBM_step(Grid grid,
                        SubgridArray subgrid_FROM_D,
                        SubgridArray subgrid_TO_D,
                        int horizontal_uncomputed_number, int vertical_uncomputed_number,
                        bool has_from_interface_horizontal,
                        bool has_from_interface_vertical,
                        bool has_to_interface_horizontal,
                        bool has_to_interface_vertical,
                        double *interface_left, double *interface_right,
                        double *interface_down, double *interface_up,
                        int subgridX, int subgridY)
{
    int stride = blockDim.x * gridDim.x;

    int cellNum = grid.subgridTrueSize[0] * grid.subgridTrueSize[1];

    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < cellNum; id += stride)
    {
        int subgrid_true_x = id % grid.subgridTrueSize[0];
        int subgrid_true_y = id / grid.subgridTrueSize[0];

        if(subgrid_true_x < horizontal_uncomputed_number || subgrid_true_x >= grid.subgridTrueSize[0] - horizontal_uncomputed_number ||
           subgrid_true_y < vertical_uncomputed_number || subgrid_true_y >= grid.subgridTrueSize[1] - vertical_uncomputed_number)
        {
            continue;
        }

        double f[3][3];

        // shift
        for(int d=0; d<grid.directionsNumber; d++)
        {
            double *target_FROM_subgrid = subgrid_FROM_D.subgrid[d];
            int target_true_x = subgrid_true_x - get_dir(d,0);
            int target_true_y = subgrid_true_y - get_dir(d,1);
            for(int c=0; c<grid.conservativesNumber; c++)
            {
                if(has_from_interface_vertical && target_true_y < grid.overlapSize[1])
                { // Read from the down interface
                    assert(target_true_y == 0);
                    assert(grid.overlapSize[1] == 1);
                    f[d][c] = interface_down[c*grid.subgridTrueSize[0] + target_true_x];
                    //printf("%d %d %d %d reads value %f from down interface\n", subgrid_true_x, subgrid_true_y, c, d, f[d][c]);
                }
                else if(has_from_interface_vertical && target_true_y >= grid.subgridTrueSize[1] - grid.overlapSize[1])
                { // Read from the up interface
                    assert(target_true_y == grid.subgridTrueSize[1] - 1);
                    assert(grid.overlapSize[1] == 1);
                    f[d][c] = interface_up[c*grid.subgridTrueSize[0] + target_true_x];
                    //printf("%d %d %d %d reads value %f from up interface\n", subgrid_true_x, subgrid_true_y, c, d, f[d][c]);
                }
                else if(has_from_interface_horizontal && target_true_x < grid.overlapSize[0])
                { // Read from the left interface
                    f[d][c] = interface_left[
                        d*grid.overlapSize[0]*grid.subgridTrueSize[1]*grid.conservativesNumber +
                        c*grid.overlapSize[0]*grid.subgridTrueSize[1] +
                        target_true_y*grid.overlapSize[0] +
                        target_true_x];
                    //printf("%d %d %d %d reads value %f from left interface\n", subgrid_true_x, subgrid_true_y, c, d, f[d][c]);
                }
                else if(has_from_interface_horizontal && target_true_x >= grid.subgridTrueSize[0] - grid.overlapSize[1])
                { // Read from the right interface
                    f[d][c] = interface_right[
                        d*grid.overlapSize[0]*grid.subgridTrueSize[1]*grid.conservativesNumber +
                        c*grid.overlapSize[0]*grid.subgridTrueSize[1] +
                        target_true_y*grid.overlapSize[0] +
                        target_true_x];
                    //printf("%d %d %d %d reads value %f from right interface\n", subgrid_true_x, subgrid_true_y, c, d, f[d][c]);
                }
                else
                { // Main case: in the logical space
                    f[d][c] = target_FROM_subgrid[c*grid.subgridTrueSize[0]*grid.subgridTrueSize[1] + target_true_y * grid.subgridTrueSize[0] + target_true_x];
                }
            }
        }

        // relax
        double w[3];
        kin_to_fluid(&f[0][0], w);
        double feq[3][3];
        fluid_to_kin(w, &feq[0][0]);
        for(int d=0; d<grid.directionsNumber; d++)
        {
            for(int c=0; c<grid.conservativesNumber; c++)
            {
                f[d][c] = OMEGA_RELAX*feq[d][c] + (1.0 - OMEGA_RELAX)*f[d][c];
            }
        }

        int position_in_interface_left_x = subgrid_true_x - grid.overlapSize[0];
        int position_in_interface_left_y = subgrid_true_y - grid.overlapSize[1];
        int position_in_interface_right_x = subgrid_true_x - grid.subgridOwnedSize[0];
        int position_in_interface_right_y = subgrid_true_y - grid.overlapSize[1];

        int position_in_interface_down_x = subgrid_true_x - grid.overlapSize[0];
        int position_in_interface_down_y = subgrid_true_y - grid.overlapSize[1];
        int position_in_interface_up_x = subgrid_true_x - grid.overlapSize[0];
        int position_in_interface_up_y = subgrid_true_y - grid.subgridOwnedSize[1];

        for(int d=0; d<grid.directionsNumber; d++)
        {
            double *target_TO_subgrid = subgrid_TO_D.subgrid[d];
            for(int c=0; c<grid.conservativesNumber; c++)
            {
                /*if(has_to_interface_vertical && position_in_interface_down_x >= 0 && position_in_interface_down_x < grid.overlapSize[1])
                { // Write to the up interface
                    assert(position_in_interface_down_y == 0);
                    assert(grid.overlapSize[1] == 1);
                    interface_down[c*grid.subgridTrueSize[0] + position_in_interface_down_x] = f[d][c];
                }
                if(has_to_interface_vertical && position_in_interface_up_x >= 0 && position_in_interface_up_x < grid.overlapSize[1])
                { // Write to the down interface
                    assert(position_in_interface_up_y == 0);
                    assert(grid.overlapSize[1] == 1);
                    interface_up[c*grid.subgridTrueSize[0] + position_in_interface_up_x] = f[d][c];
                }
                if(has_to_interface_horizontal && position_in_interface_left_y >= 0 && position_in_interface_left_y < grid.overlapSize[1])
                { // Write to the left interface
                    interface_left[c*grid.overlapSize[0]*grid.subgridTrueSize[1] + position_in_interface_left_y*grid.overlapSize[0] + position_in_interface_left_x] = f[d][c];
                }
                if(has_to_interface_horizontal && position_in_interface_right_y >= 0 && position_in_interface_right_y < grid.overlapSize[1])
                { // Write to the right interface
                    interface_right[c*grid.overlapSize[0]*grid.subgridTrueSize[1] + position_in_interface_right_y*grid.overlapSize[0] + position_in_interface_right_x] = f[d][c];
                }*/

                if(has_to_interface_vertical && position_in_interface_down_y >= 0 && position_in_interface_down_y < grid.overlapSize[1])
                {
                    assert(grid.overlapSize[1] == 1);
                    interface_down[c*grid.overlapSize[0] + position_in_interface_down_x] = f[d][c];
                }
                if(has_to_interface_vertical && position_in_interface_up_y >= 0 && position_in_interface_up_y < grid.overlapSize[1])
                {
                    assert(grid.overlapSize[1] == 1);
                    interface_up[c*grid.overlapSize[0] + position_in_interface_up_x] = f[d][c];
                }
                if(has_to_interface_horizontal && position_in_interface_left_x >= 0 && position_in_interface_left_x < grid.overlapSize[0])
                {
                    interface_left[d*grid.overlapSize[0]*grid.subgridTrueSize[1]*grid.conservativesNumber +
                            c*grid.overlapSize[0]*grid.subgridTrueSize[1] +
                            position_in_interface_left_y*grid.overlapSize[0] +
                            position_in_interface_left_x]
                        = f[d][c];
                }
                if(has_to_interface_horizontal && position_in_interface_right_x >= 0 && position_in_interface_right_x < grid.overlapSize[0])
                {
                    interface_right[d*grid.overlapSize[0]*grid.subgridTrueSize[1]*grid.conservativesNumber +
                            c*grid.overlapSize[0]*grid.subgridTrueSize[1] +
                            position_in_interface_left_y*grid.overlapSize[0] +
                            position_in_interface_left_x]
                        = f[d][c];
                }
                
                // In all cases, write to the subgrid
                target_TO_subgrid[c*grid.subgridTrueSize[0]*grid.subgridTrueSize[1] + subgrid_true_y * grid.subgridTrueSize[0] + subgrid_true_x] = f[d][c];
            }
        }
    }
}
